#include <iostream>
#include "error_handler.hpp"

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>

int main()
{
    // error: invalid value
    double *d = 0;
    double *a = 0;
    error_check(hipMemcpy(d, a, sizeof(int), hipMemcpyHostToDevice));

    return 0;
}
